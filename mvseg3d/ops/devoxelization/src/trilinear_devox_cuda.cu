#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <cmath>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define MAXIMUM_THREADS 512

inline int optimal_num_threads(int work_size) {
  const int pow_2 = std::log2(static_cast<double>(work_size));
  return max(min(1 << pow_2, MAXIMUM_THREADS), 1);
}

/*
  Function: trilinear devoxlization (forward)
  Args:
    b   : batch size
    c   : channels
    l   : voxel length
    w   : voxel width
    h   : voxel height
    n   : number of points
    coords : the coordinates of points, FloatTensor[b, 3, n]
    feat   : features, FloatTensor[b, c, r3]
    inds   : the voxel indices of point cube, IntTensor[b, 8, n]
    wgts   : weight for trilinear interpolation, FloatTensor[b, 8, n]
    outs   : outputs, FloatTensor[b, c, n]
*/
__global__ void trilinear_devoxelize_kernel(int b, int c, int l, int w, int h,
                                            int n, bool is_training,
                                            const float *__restrict__ coords,
                                            const float *__restrict__ feat,
                                            int *__restrict__ inds,
                                            float *__restrict__ wgts,
                                            float *__restrict__ outs) {
  int batch_index = blockIdx.x;
  int stride = blockDim.x;
  int index = threadIdx.x;
  coords += batch_index * n * 3;
  inds += batch_index * n * 8;
  wgts += batch_index * n * 8;
  feat += batch_index * c * l * w * h;
  outs += batch_index * c * n;

  for (int i = index; i < n; i += stride) {
    float x = coords[i];
    float y = coords[i + n];
    float z = coords[i + n + n];
    if (x < 0 || x >= l || y < 0 || y >= w || z < 0 || z >= h) {
        continue;
    }
    float x_lo_f = floorf(x);
    float y_lo_f = floorf(y);
    float z_lo_f = floorf(z);

    float x_d_1 = x - x_lo_f; // / (x_hi_f - x_lo_f + 1e-8f)
    float y_d_1 = y - y_lo_f;
    float z_d_1 = z - z_lo_f;
    float x_d_0 = 1.0f - x_d_1;
    float y_d_0 = 1.0f - y_d_1;
    float z_d_0 = 1.0f - z_d_1;

    float wgt000 = x_d_0 * y_d_0 * z_d_0;
    float wgt001 = x_d_0 * y_d_0 * z_d_1;
    float wgt010 = x_d_0 * y_d_1 * z_d_0;
    float wgt011 = x_d_0 * y_d_1 * z_d_1;
    float wgt100 = x_d_1 * y_d_0 * z_d_0;
    float wgt101 = x_d_1 * y_d_0 * z_d_1;
    float wgt110 = x_d_1 * y_d_1 * z_d_0;
    float wgt111 = x_d_1 * y_d_1 * z_d_1;

    int x_lo = static_cast<int>(x_lo_f);
    int y_lo = static_cast<int>(y_lo_f);
    int z_lo = static_cast<int>(z_lo_f);
    int x_hi = (x_d_1 > 0) ? -1 : 0;
    int y_hi = (y_d_1 > 0) ? -1 : 0;
    int z_hi = (z_d_1 > 0) ? 1 : 0;

    int idx000 = x_lo * w * h + y_lo * h + z_lo;
    int idx001 = idx000 + z_hi;      // x_lo * w * h + y_lo * h + z_hi;
    int idx010 = idx000 + (y_hi & h);  // x_lo * w * h + y_hi * h + z_lo;
    int idx011 = idx010 + z_hi;      // x_lo * w * h + y_hi * h + z_hi;
    int idx100 = idx000 + (x_hi & (w * h)); // x_hi * w * h + y_lo * h + z_lo;
    int idx101 = idx100 + z_hi;      // x_hi * w * h + y_lo * h + z_hi;
    int idx110 = idx100 + (y_hi & h);  // x_hi * w * h + y_hi * h + z_lo;
    int idx111 = idx110 + z_hi;      // x_hi * w * h + y_hi * h + z_hi;

    if (is_training) {
      wgts[i] = wgt000;
      wgts[i + n] = wgt001;
      wgts[i + n * 2] = wgt010;
      wgts[i + n * 3] = wgt011;
      wgts[i + n * 4] = wgt100;
      wgts[i + n * 5] = wgt101;
      wgts[i + n * 6] = wgt110;
      wgts[i + n * 7] = wgt111;
      inds[i] = idx000;
      inds[i + n] = idx001;
      inds[i + n * 2] = idx010;
      inds[i + n * 3] = idx011;
      inds[i + n * 4] = idx100;
      inds[i + n * 5] = idx101;
      inds[i + n * 6] = idx110;
      inds[i + n * 7] = idx111;
    }

    for (int j = 0; j < c; j++) {
      int j_size = j * l * w * h;
      outs[j * n + i] =
          wgt000 * feat[j_size + idx000] + wgt001 * feat[j_size + idx001] +
          wgt010 * feat[j_size + idx010] + wgt011 * feat[j_size + idx011] +
          wgt100 * feat[j_size + idx100] + wgt101 * feat[j_size + idx101] +
          wgt110 * feat[j_size + idx110] + wgt111 * feat[j_size + idx111];
    }
  }
}

/*
  Function: trilinear devoxlization (backward)
  Args:
    b      : batch size
    c      : channels
    size   : voxel cube size = l*w*h
    n      : number of points
    inds   : the voxel indices of point cube, IntTensor[b, 8, n]
    wgts   : weight for trilinear interpolation, FloatTensor[b, 8, n]
    grad_y : grad outputs, FloatTensor[b, c, n]
    grad_x : grad inputs, FloatTensor[b, c, r3]
*/
__global__ void trilinear_devoxelize_grad_kernel(
    int b, int c, int size, int n, const int *__restrict__ inds,
    const float *__restrict__ wgts, const float *__restrict__ grad_y,
    float *__restrict__ grad_x) {
  int batch_index = blockIdx.x;
  int stride = blockDim.x;
  int index = threadIdx.x;
  inds += batch_index * n * 8;
  wgts += batch_index * n * 8;
  grad_x += batch_index * c * size;
  grad_y += batch_index * c * n;

  for (int i = index; i < n; i += stride) {
    int idx000 = inds[i];
    int idx001 = inds[i + n];
    int idx010 = inds[i + n * 2];
    int idx011 = inds[i + n * 3];
    int idx100 = inds[i + n * 4];
    int idx101 = inds[i + n * 5];
    int idx110 = inds[i + n * 6];
    int idx111 = inds[i + n * 7];
    float wgt000 = wgts[i];
    float wgt001 = wgts[i + n];
    float wgt010 = wgts[i + n * 2];
    float wgt011 = wgts[i + n * 3];
    float wgt100 = wgts[i + n * 4];
    float wgt101 = wgts[i + n * 5];
    float wgt110 = wgts[i + n * 6];
    float wgt111 = wgts[i + n * 7];

    for (int j = 0; j < c; j++) {
      int j_size = j * size;
      float g = grad_y[j * n + i];
      atomicAdd(grad_x + j_size + idx000, wgt000 * g);
      atomicAdd(grad_x + j_size + idx001, wgt001 * g);
      atomicAdd(grad_x + j_size + idx010, wgt010 * g);
      atomicAdd(grad_x + j_size + idx011, wgt011 * g);
      atomicAdd(grad_x + j_size + idx100, wgt100 * g);
      atomicAdd(grad_x + j_size + idx101, wgt101 * g);
      atomicAdd(grad_x + j_size + idx110, wgt110 * g);
      atomicAdd(grad_x + j_size + idx111, wgt111 * g);
    }
  }
}

void trilinear_devoxelize(int b, int c, int l, int w, int h, int n,
                          bool training, const float *coords, const float *feat,
                          int *inds, float *wgts, float *outs) {
  trilinear_devoxelize_kernel<<<b, optimal_num_threads(n)>>>(
      b, c, l, w, h, n, training, coords, feat, inds, wgts, outs);
}

void trilinear_devoxelize_grad(int b, int c, int size, int n, const int *inds,
                               const float *wgts, const float *grad_y,
                               float *grad_x) {
  trilinear_devoxelize_grad_kernel<<<b, optimal_num_threads(n)>>>(
      b, c, size, n, inds, wgts, grad_y, grad_x);
}